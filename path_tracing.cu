#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "path_tracing.h"
#include "ray.h"
#include "sphere.h"
#include "hittable.h"
#include "stdio.h"

__global__ void init_world(Hittable** dev_world)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        dev_world[0] = new Sphere({ 0, 0, -1 }, 0.5);
        dev_world[1] = new Sphere({ 0, -100.5, -1 }, 100);
    }
}

__device__ vec3 color(Ray* ray, Hittable** dev_world, int world_size)
{
    HitRecord rec;
    float t_min = 0.0, t_max = 1E9;
    bool hit_anything = false;
    float closest_so_far = t_max;
    for (int i = 0; i < world_size; i++)
    {
        if (dev_world[i]->hit(ray, t_min, closest_so_far, &rec))
        {
            hit_anything = true;
            closest_so_far = rec.t;
        }
    }
    if(hit_anything)
    {
        vec3 ones = { 1, 1, 1 };
        return (rec.normal + ones) * 0.5;
    }
    vec3 unit_direction = ray->direction.unit_vector();
    float t = 0.5 * (unit_direction.Y + 1.0);
    return {
        1.0 - t + 0.5 * t,
        1.0 - t + 0.7 * t,
        1.0 - t + 1.0 * t
    };
}

__global__ void path_tracing_kernel(Hittable** dev_world, int world_size, vec3* dev_framebuffer, int height, int width)
{
    int size = width * height;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        int i = idx % width;
        int j = idx / width;
        // camera
        vec3 lower_left_corner = { -2, -1, -1 };
        vec3 horizontal = { 4, 0, 0 };
        vec3 vertical = { 0, 2, 0 };
        vec3 origin = { 0, 0, 0 };
        //ray
        float u = float(i) / float(width);
        float v = float(j) / float(height);
        Ray ray = {
            origin, 
            lower_left_corner + horizontal * u + vertical * v
        };
        dev_framebuffer[idx] = color(&ray, dev_world, world_size);
    }
}

hipError_t path_tracing_with_cuda(vec3* framebuffer, int height, int width)
{
    hipError_t cudaStatus = hipSetDevice(0);
    //framebuffer
    int size = width * height;
    vec3* dev_framebuffer = 0;
    cudaStatus = hipMalloc((void**)&dev_framebuffer, size * sizeof(vec3));
    //world
    int world_size = 2;
    Hittable** dev_world = 0;
    cudaStatus = hipMalloc(&dev_world, world_size * sizeof(Hittable**));
    init_world <<<1,1>>>(dev_world);
    //tracing
    path_tracing_kernel <<<size / 512 + 1, 512>>>(dev_world, world_size, dev_framebuffer, height, width);
    cudaStatus = hipGetLastError();
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(framebuffer, dev_framebuffer, size * sizeof(vec3), hipMemcpyDeviceToHost);
    hipFree(dev_framebuffer);
    hipFree(dev_world);
    return cudaStatus;
}
