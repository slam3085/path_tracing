#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "path_tracing.h"
#include "ray.h"
#include "sphere.h"
#include "hittable.h"
#include "hittable_list.h"

__global__ void init_world(Hittable** dev_world)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        Hittable** list = new Hittable*[2];
        list[0] = new Sphere({ 0, 0, -1 }, 0.5);
        list[1] = new Sphere({ 0, -100.5, -1 }, 100);
        *dev_world = new HittableList(list, 2);
    }
}

__device__ vec3 color(Ray* ray, Hittable** dev_world)
{
    HitRecord rec;
    if((*dev_world)->hit(ray, 0.0, 1E9, &rec))
    {
        vec3 ones = { 1, 1, 1 };
        return (rec.normal + ones) * 0.5;
    }
    vec3 unit_direction = ray->direction.unit_vector();
    float t = 0.5 * (unit_direction.Y + 1.0);
    return {
        1.0 - t + 0.5 * t,
        1.0 - t + 0.7 * t,
        1.0 - t + 1.0 * t
    };
}

__global__ void path_tracing_kernel(Hittable** dev_world, vec3* dev_framebuffer, int height, int width)
{
    int size = width * height;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        int i = idx % width;
        int j = idx / width;
        // camera
        vec3 lower_left_corner = { -2, -1, -1 };
        vec3 horizontal = { 4, 0, 0 };
        vec3 vertical = { 0, 2, 0 };
        vec3 origin = { 0, 0, 0 };
        //ray
        float u = float(i) / float(width);
        float v = float(j) / float(height);
        Ray ray = {
            origin, 
            lower_left_corner + horizontal * u + vertical * v
        };
        dev_framebuffer[idx] = color(&ray, dev_world);
    }
}

hipError_t path_tracing_with_cuda(vec3* framebuffer, int height, int width)
{
    hipError_t cudaStatus = hipSetDevice(0);
    //framebuffer
    int size = width * height;
    vec3* dev_framebuffer = 0;
    cudaStatus = hipMalloc((void**)&dev_framebuffer, size * sizeof(vec3));
    //world
    Hittable** dev_world = 0;
    cudaStatus = hipMalloc(&dev_world, sizeof(Hittable**));
    init_world <<<1,1>>>(dev_world);
    //tracing
    path_tracing_kernel <<<size / 512 + 1, 512>>>(dev_world, dev_framebuffer, height, width);
    cudaStatus = hipGetLastError();
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(framebuffer, dev_framebuffer, size * sizeof(vec3), hipMemcpyDeviceToHost);
    hipFree(dev_framebuffer);
    hipFree(dev_world);
    return cudaStatus;
}
