#include "hip/hip_runtime.h"
#include ""
#include "ray.h"
#include "vec3.h"

__device__ vec3 Ray::point_at_parameter(float t)
{
    return origin + direction * t;
}