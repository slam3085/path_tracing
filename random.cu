#include "hip/hip_runtime.h"
#include "random.h"

__device__ float random_float(hiprandState_t* state)
{
    //hiprand_uniform returns values from (0.0, 1.0]
    return 1.0f - hiprand_uniform(state);
}

__device__ vec3 random_unit_in_sphere(hiprandState_t* state)
{
    vec3 p(2.0f * random_float(state) - 1.0f, 2.0f * random_float(state) - 1.0f, 2.0f * random_float(state) - 1.0f);
    float a = 2.0f * random_float(state) - 1.0f, b = 2.0f * random_float(state) - 1.0f;
    float l = sqrtf(p.squared_length() + a * a + b * b);
    p /= l;
    return p;
}