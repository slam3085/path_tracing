#include "random.h"

__device__ float random_float(hiprandState_t* state)
{
    //hiprand_uniform returns values from (0.0, 1.0]
    return 1.0f - hiprand_uniform(state);
}

__device__ vec3 random_unit_in_sphere(hiprandState_t* state)
{
    vec3 p;
    while(true)
    {
        p.X = 2.0f * random_float(state) - 1.0f;
        p.Y = 2.0f * random_float(state) - 1.0f;
        p.Z = 2.0f * random_float(state) - 1.0f;
        if(p.squared_length() < 1.0f)
            return p;
    }
}