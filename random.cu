#include "random.h"


__device__ float random_float(hiprandState_t* state)
{
    return float(hiprand(state) % 10000) / 10001.0;
}