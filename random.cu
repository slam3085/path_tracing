#include "random.h"

__device__ float random_float(hiprandState_t* state)
{
    //hiprand_uniform returns values from (0.0, 1.0]
    float tmp = hiprand_uniform(state);
    if(tmp >= 1.0f)
        return 0.0f;
    return tmp;
}

__device__ vec3 random_unit_in_sphere(hiprandState_t* state)
{
    vec3 p;
    while(true)
    {
        p.X = 2.0f * random_float(state) - 1.0f;
        p.Y = 2.0f * random_float(state) - 1.0f;
        p.Z = 2.0f * random_float(state) - 1.0f;
        if(p.squared_length() < 1.0f)
            return p;
    }
}