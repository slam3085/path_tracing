#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gradient.h"

__global__ void gradientKernel(float* dev_framebuffer, int height, int width)
{
	int size = 3 * width * height;
	int i = 3 * (blockIdx.x * blockDim.x + threadIdx.x);
	if (i < size)
	{
		dev_framebuffer[i] = float(threadIdx.x) / float(width);
		dev_framebuffer[i + 1] = float(blockIdx.x) / float(height);
		dev_framebuffer[i + 2] = 0.2;
	}
}

hipError_t gradientWithCuda(float* framebuffer, int height, int width)
{
	int size = 3 * width * height;
	hipError_t cudaStatus = hipSetDevice(0);
	float* dev_framebuffer = 0;
	cudaStatus = hipMalloc((void**)&dev_framebuffer, size * sizeof(float));
    cudaStatus = hipMemcpy(dev_framebuffer, framebuffer, size * sizeof(float), hipMemcpyHostToDevice);
	gradientKernel <<<height, width >>>(dev_framebuffer, height, width);
    cudaStatus = hipGetLastError();
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(framebuffer, dev_framebuffer, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_framebuffer);
    return cudaStatus;
}
