#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include "gradient.h"
#include "ray.h"

__device__ vec3 color(Ray* ray)
{
    vec3 unit_direction = ray->direction.unit_vector();
    float t = 0.5 * unit_direction.Y + 1.0;
    return {
        1.0 - t + 0.5 * t,
        1.0 - t + 0.7 * t,
        1.0 - t + 1.0 * t
    };
}

__global__ void gradientKernel(vec3* dev_framebuffer, int height, int width)
{
    int size = width * height;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        vec3 lower_left_corner = { -2, -1, -1 };
        vec3 horizontal = { 4, 0, 0 };
        vec3 vertical = { 0, 2, 0 };
        vec3 origin = { 0, 0, 0 };
        float u = float(threadIdx.x) / float(width);
        float v = float(blockIdx.x) / float(height);
        Ray ray = {
            origin, 
            lower_left_corner + horizontal * u + vertical * v
        };
        dev_framebuffer[i] = color(&ray);
    }
}

hipError_t gradientWithCuda(vec3* framebuffer, int height, int width)
{
    int size = width * height;
    hipError_t cudaStatus = hipSetDevice(0);
    vec3* dev_framebuffer = 0;
    cudaStatus = hipMalloc((void**)&dev_framebuffer, size * sizeof(vec3));
    cudaStatus = hipMemcpy(dev_framebuffer, framebuffer, size * sizeof(vec3), hipMemcpyHostToDevice);
    gradientKernel <<<height, width >>>(dev_framebuffer, height, width);
    cudaStatus = hipGetLastError();
    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(framebuffer, dev_framebuffer, size * sizeof(vec3), hipMemcpyDeviceToHost);
    hipFree(dev_framebuffer);
    return cudaStatus;
}
